﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size, int* sum);

__global__ void addKernel(int size, int* c, const int* a, const int* b, int* sum)
{
    int i = threadIdx.x;
    while (i < size) {
        c[i] = a[i] + b[i];
        atomicAdd(sum, a[i]);
        i += (gridDim.x * blockDim.x) * (gridDim.y * blockDim.y);
    }
}

int main()
{
    const int arraySize = 10000;
    int* sum = new int;
    int* a = new int[arraySize];
    int* b = new int[arraySize];
    int* c = new int[arraySize];

    for (int i = 0; i < arraySize; i++) {
        a[i] = i + 1;
        b[i] = (i + 1) * 10;
    }

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize, sum);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    //printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        //c[0], c[1], c[2], c[3], c[4]);

    std::cout << *sum << std::endl;
    for (int i = 0; i < arraySize; i++)
        std::cout << c[i] << ' ';
    std::cout << std::endl;

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size, int* sum)
{
    int *dev_sum;
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_sum, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemset(dev_sum, 0, sizeof(int));

    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, 1000>>>(size, dev_c, dev_a, dev_b, dev_sum);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(sum, dev_sum, sizeof(int), hipMemcpyDeviceToHost);
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
