﻿#include "hip/hip_runtime.h"
#include ""

#include <cassert>
#include <cstdio>
#include <iostream>



int main()
{
    int size = 10000;
    int* a;
    int* aGPU;
    int* locks;

    a = new int[size];

    hipMalloc(&aGPU, sizeof(int) * size);
    hipMemset(aGPU, 0, sizeof(int) * size);

    hipMalloc(&locks, sizeof(int) * size);
    hipMemset(locks, 0, sizeof(int) * size);

    maxKernel<<<64, 64>>>(size, aGPU, locks);
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));

    hipMemcpy(a, aGPU, sizeof(int) * size, hipMemcpyDeviceToHost);
    for (int i = 0; i < size - 1; i++)
        std::cout << a[i] << ' ';
    std::cout << a[size - 1] << std::endl;


    delete[] a;
    hipFree(aGPU);
    hipFree(locks);

    return 0;
}
