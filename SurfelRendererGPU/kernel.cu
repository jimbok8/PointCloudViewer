#include "hip/hip_runtime.h"
﻿#include <cassert>
#include <iostream>

#include <hip/hip_runtime.h>
#include <>

#include <GLFW/glfw3.h>

#include "TypeHelper.h"
#include "Vector3D.h"

void surfaceSplatStep1(int width, int height, ZBufferProperty* zBufferProperty, ZBufferItem* zBuffer, float* filterLUT, float x0, float y0, float z, float n[3], Surfel* surfel, int l, float scale_stoo, float scale_otoc, float vp_sx, float vp_sy, float vp_tx, float vp_ty, int bbox[4])
{
	float zbf_LUTsize = zBufferProperty->LUTsize;
	float zbf_cutoffRadius_2 = zBufferProperty->cutoffRadius * zBufferProperty->cutoffRadius;
	float _zbf_cutoffRadius_2 = 1 / (zbf_cutoffRadius_2);
	float zbf_angleThreshold = zBufferProperty->angleTrheshold;

	float V_x, V_y, V_z;		// viewing direction
	float S_x, S_y, S_z;		// S parameter direction on ST plane
	float T_x, T_y, T_z;		// T parameter direction on ST plane
	float Ix_x, Ix_y, Ix_z;		// direction of projection of screen x vector onto ST plane
	float Iy_x, Iy_y, Iy_z;		// direction of projection of screen y vector onto ST plane
	float r, r_, f;
	float ndotv;				// N*V (normal dot viewing direction) dotproduct
	float sx, sy, tx, ty;		// derivatives of the screen to ST plane mapping
	float m11, m12, m22;
	float a, b, b_2, c;			// the EWA ellipse coefficients

	int Xmin, Xmax, Ymin, Ymax;	// bounding box of the ellipse to be rasterized
	float lx, ly;
	int X, Y;
	float x, y;
	float q, dq, ddq;

	float dzc_dxs, dzc_dys;			// dzc/dxs, dzc/dys derivatives
	float z_start, z_cur;			// z values to be rasterized

	int i;
	float r_comp, g_comp, b_comp;

	float threshold_c3;
	float e;
	float w;

	float l_2, _l_2;
	float det_;

	Vector3D normal;
	float _radius;

	//scale z buffer according to surfel radius
	float scale_otoc_radius;

	// init bounding box return value
	bbox[0] = width;
	bbox[1] = height;
	bbox[2] = bbox[3] = 0;

	l_2 = (float)(1 << l);
	_l_2 = 1 / l_2;

	// thresholds for the 'merge/separate' decision
	// note: 
	// - these thresholds are dependent on the level of the warped samples, since the
	// distance on the surface between samples increases with the warped level.
	// - these offsets should be constant in units 'dz per unit in object space'. but since 
	// z coordinates in the z-buffer are in camera coordinates, we have to transform the offsets
	// to camera space by applying the 'scale_otoc' (object to camera space) scaling
	threshold_c3 = zbf_angleThreshold;							// cut off for 'ndotv' used for calculating the derivatives dzc_dxs,dzc_dys

	l_2 *= l_2;

	// step 1: calculate the EWA ellipse coefficients

	// compute normalized viewing vector V
	// restore camera coordinates of projected point (on z=1 plane)
	V_x = -(x0 / vp_sx - vp_tx);
	V_y = -(y0 / vp_sy - vp_ty);
	V_z = -1.f;
	r_ = 1.f / (float)sqrt(V_x * V_x + V_y * V_y + 1.f);
	V_x *= r_; V_y *= r_; V_z *= r_;
	ndotv = n[0] * V_x + n[1] * V_y + n[2] * V_z;

	if (ndotv < 0) {
		n[0] = -n[0];
		n[1] = -n[1];
		n[2] = -n[2];
		ndotv = -ndotv;
	}

	// project screen x,y unit vectors along V onto ST plane
	// determine direction Ix,Iy of projection of x,y
	Ix_x = -V_z * n[2] - V_y * n[1];		// (X x V) x N
	Ix_y = V_y * n[0];
	Ix_z = V_z * n[0];
	Iy_x = V_x * n[1];				// (Y x V) x N
	Iy_y = -V_x * n[0] - V_z * n[2];
	Iy_z = V_z * n[1];

	// f given by the vector equation Y+g*V=f*Iy resp X+g*V=f*Ix
	// Iy*f resp. Ix*f is the intersection point of Iy,Ix with Y+g*V resp. X+g*V
	// (see SurfaceSplatting.mws)
	f = -1.f / ndotv;

	// dzdx, dzdy derivatives for rasterizing z values on the ellipse
	if (f < -threshold_c3) e = -threshold_c3; else e = f;
	// note: since z coordinates in the z buffer are camera coordinate z-values, we have
	// to transform from screen to camera coordinates (by concatenating the screen-to-object
	// and object-to-camera scalings).
	dzc_dxs = Ix_z * e * scale_stoo * scale_otoc * z;
	dzc_dys = Iy_z * e * scale_stoo * scale_otoc * z;
	e = -e;

	// normalize Ix
	r = (float)sqrt(Ix_x * Ix_x + Ix_y * Ix_y + Ix_z * Ix_z);
	r_ = 1 / r;
	S_x = Ix_x * r_;
	S_y = Ix_y * r_;
	S_z = Ix_z * r_;

	// determine T parameter direction on ST plane
	// note: S parameter direction is chosen as Ix
	// T is automatically normalized,  since n and S have unit length
	// and are orthogonal by construction of S
	T_x = n[1] * S_z - n[2] * S_y;		// N x S
	T_y = n[2] * S_x - n[0] * S_z;
	T_z = n[0] * S_y - n[1] * S_x;

	// compute sx, sy, tx, ty derivatives
	// these derivatives build the inverse jacobian inv(J)=[sx,sy | tx,ty]
	// of the mapping J from object surface to screen

	_radius = surfel->radius;
	scale_otoc_radius = scale_otoc * _radius;
	_radius = 1.0f / _radius;

	Iy_x *= f; Iy_y *= f; Iy_z *= f;		// f*Iy is the intersection point with Y+g*V
	sx = r * f * scale_stoo * z * _radius * _l_2;		// note f given by the vector equation X+g*V=f*Ix takes the same value as above
	tx = 0;
	sy = (Iy_x * S_x + Iy_y * S_y + Iy_z * S_z) * scale_stoo * z * _radius * _l_2;		// Iy*S projects Iy onto S
	ty = (Iy_x * T_x + Iy_y * T_y + Iy_z * T_z) * scale_stoo * z * _radius * _l_2;		// Iy*T projects Iy onto T

	// compute z-range of the reconstruction kernel
	// see ellipseboundingbox_general.mws, idea: compute the point on the ellipse, where the ellipse tangent
	// is perpendicular to the depth gradient, i.e. [dzc_dxs, dzc_dys]. this is the point with maximum depth
	// on the ellipse
	// NOTE: to avoid overshoots in z, we use the z-range of the reconstruction kernel for blending!
	// NOTE: the variable "d" in the maple sheet corresponds to "-zbf_cutoffRadius_2"!
	// first, compute the conic matrix of the reconstruction kernel, which is [a b/2 | b/2 c] = inv(J)^T*inv(J)
	a = sx * sx;
	b_2 = sx * sy;
	b = 2.f * b_2;
	c = sy * sy + ty * ty;
	float discr;
	discr = -4 * a * dzc_dxs * b * dzc_dys * c - a * dzc_dys * dzc_dys * b * b + 4 * c * dzc_dys * dzc_dys * a * a +
		4 * a * dzc_dxs * dzc_dxs * c * c + b * b * b * dzc_dxs * dzc_dys - b * b * dzc_dxs * dzc_dxs * c;

	float zExtremum_x, zExtremum_y, tmp;
	tmp = (float)sqrt(discr * zbf_cutoffRadius_2);
	zExtremum_x = tmp * (-dzc_dys * b + 2 * dzc_dxs * c) / discr;
	zExtremum_y = tmp * (-2 * dzc_dys * a + dzc_dxs * b) / discr;

	float zRange_x, zRange_y;
	tmp = zExtremum_x * dzc_dxs;
	zRange_x = (tmp < 0) ? -tmp : tmp;
	tmp = zExtremum_y * dzc_dys;
	zRange_y = (tmp < 0) ? -tmp : tmp;

	float zMax, zMin;
	zMax = z + zRange_x + zRange_y;
	zMin = z - zRange_x - zRange_y;

	// guarantee a certain minimum z-range, otherwise blending fails for splats exactly parallel to the
	// image plane (the minimum z-range is 1 in object space, and then scaled to camera space).
	if (zMax - zMin < l_2 * scale_otoc_radius) {
		zMax += 0.5f * l_2 * scale_otoc_radius;
		zMin -= 0.5f * l_2 * scale_otoc_radius;
	}

	// calculate the matrix e[]=inv(J*transpose(J)+I), which describes the
	// EWA ellipse on the screen
	// note: column vectors are used, therefore the calculations are slightly
	// different to Heckbert's notation (cf. Maple file SurfaceSplatting2.mws)

	// the filtered variance matrix m[]
	// m[] = J*transpose(J)+I (details see SurfaceSplatting2.mws)
	r_ = 1.f / (sx * ty);
	r_ *= r_;							// r_ = 1/(sx*ty)^2, note that always tx=0, therefore some terms are missing
	m11 = (ty * ty + sy * sy) * r_ + 1.0f;	// in these formulas compared to the maple sheet
	m12 = -sy * sx * r_;
	m22 = sx * sx * r_ + 1.0f;

	// the filtered conic matric m[]^(-1)
	// matrix(A,B/2,B/2,C) = inv(m[])
	r_ = 1.f / (m11 * m22 - m12 * m12);
	a = m22 * r_;
	b_2 = -m12 * r_;
	b = b_2 * 2.f;
	c = m11 * r_;

	// calculate the normalization factor for the gaussian filter, which corresponds to the 
	// area of the reconstruction filter in source space, thus it is 
	// 1 / sqrt( det(inv(J)*transpose(inv(J))+I) )
	// note: the factor l_2 compensate for the level of detail of the LDC tree
	det_ = l_2 / (float)sqrt(sx * sx * ty * ty + sx * sx + sy * sy + ty * ty + 1.f);

	// bounding box of the ellipse
	// see ellipseboundingbox.mws, an exact axis aligned bounding box is computed by finding the points on
	// the ellipse where the tangent of the ellipse is parallel to x- and y-axis respectively.
	// NOTE: the variable "d" in the maple sheet corresponds to "-zbf_cutoffRadius_2"!
	discr = (float)sqrt((-b * b + 4 * c * a) * zbf_cutoffRadius_2 * a);
	ly = 2.f / (-b * b + 4 * c * a) * discr;

	discr = (float)sqrt(c * (-b * b + 4 * c * a) * zbf_cutoffRadius_2);
	lx = 2.f / (-b * b + 4 * c * a) * discr;

	lx = (lx < 0) ? -lx : lx;
	ly = (ly < 0) ? -ly : ly;
	Xmax = (int)(x0 + lx) + 1;
	Xmin = (int)(x0 - lx);
	Ymax = (int)(y0 + ly) + 1;
	Ymin = (int)(y0 - ly);

	// get surfel color
	r_comp = surfel->r;
	g_comp = surfel->g;
	b_comp = surfel->b;

	// step 2: rasterize the EWA ellipse

	// padding
	if (Xmin < 0) {
		Xmin = 0;
		if (Xmax < 0)
			return;
	}
	if (Xmax >= width) {
		Xmax = width - 1;
		if (Xmin >= width)
			return;
	}
	if (Ymin < 0) {
		Ymin = 0;
		if (Ymax < 0)
			return;
	}
	if (Ymax >= height) {
		Ymax = height - 1;
		if (Ymin >= height)
			return;
	}

	// set bounding box (the bounding box is a return value)
	bbox[0] = Xmin;
	bbox[1] = Ymin;
	bbox[2] = Xmax;
	bbox[3] = Ymax;

	// z value in the lower left corner of the rasterized area
	z_start = z - dzc_dxs * (x0 - ((float)Xmin + 0.5f)) - dzc_dys * (y0 - ((float)Ymin + 0.5f));

	x = ((float)Xmin + 0.5f) - x0;
	ddq = 2 * a;

	// *********************
	// ellipse rasterization
	// *********************
	for (Y = Ymin; Y <= Ymax; Y++)
	{
		// finite differences for ellipse rasterization
		y = ((float)Y + 0.5f) - y0;
		dq = a * (2 * x + 1) + b * y;
		q = (c * y + b * x) * y + a * x * x;

		// init z value
		z_cur = z_start;

		for (X = Xmin; X <= Xmax; X++)
		{
			i = X + width * Y;

			if (q < zbf_cutoffRadius_2)
			{
				if (zMin < zBuffer[i].zMin) {
					// new z-range does not overlap previous one, but is closer to viewer
					// update z-range
					zBuffer[i].zMin = zMin;
					zBuffer[i].zMax = zMax;
				}
			}
			q += dq;
			dq += ddq;

			z_cur += dzc_dxs;
		}
		z_start += dzc_dys;
	}
}

void surfaceSplatStep2(int width, int height, ZBufferProperty* zBufferProperty, ZBufferItem* zBuffer, float* filterLUT, float x0, float y0, float z, float n[3], Surfel* surfel, int l, float scale_stoo, float scale_otoc, float vp_sx, float vp_sy, float vp_tx, float vp_ty, int bbox[4])
{
	float zbf_LUTsize = zBufferProperty->LUTsize;
	float zbf_cutoffRadius_2 = zBufferProperty->cutoffRadius * zBufferProperty->cutoffRadius;
	float _zbf_cutoffRadius_2 = 1 / (zbf_cutoffRadius_2);
	float zbf_angleThreshold = zBufferProperty->angleTrheshold;

	float V_x, V_y, V_z;		// viewing direction
	float S_x, S_y, S_z;		// S parameter direction on ST plane
	float T_x, T_y, T_z;		// T parameter direction on ST plane
	float Ix_x, Ix_y, Ix_z;		// direction of projection of screen x vector onto ST plane
	float Iy_x, Iy_y, Iy_z;		// direction of projection of screen y vector onto ST plane
	float r, r_, f;
	float ndotv;				// N*V (normal dot viewing direction) dotproduct
	float sx, sy, tx, ty;		// derivatives of the screen to ST plane mapping
	float m11, m12, m22;
	float a, b, b_2, c;			// the EWA ellipse coefficients

	int Xmin, Xmax, Ymin, Ymax;	// bounding box of the ellipse to be rasterized
	float lx, ly;
	int X, Y;
	float x, y;
	float q, dq, ddq;

	float dzc_dxs, dzc_dys;			// dzc/dxs, dzc/dys derivatives
	float z_start, z_cur;			// z values to be rasterized

	int i;
	float r_comp, g_comp, b_comp;

	float threshold_c3;
	float e;
	float w;

	float l_2, _l_2;
	float det_;

	Vector3D normal;
	float _radius;

	//scale z buffer according to surfel radius
	float scale_otoc_radius;

	// init bounding box return value
	bbox[0] = width;
	bbox[1] = height;
	bbox[2] = bbox[3] = 0;

	l_2 = (float)(1 << l);
	_l_2 = 1 / l_2;

	// thresholds for the 'merge/separate' decision
	// note: 
	// - these thresholds are dependent on the level of the warped samples, since the
	// distance on the surface between samples increases with the warped level.
	// - these offsets should be constant in units 'dz per unit in object space'. but since 
	// z coordinates in the z-buffer are in camera coordinates, we have to transform the offsets
	// to camera space by applying the 'scale_otoc' (object to camera space) scaling
	threshold_c3 = zbf_angleThreshold;							// cut off for 'ndotv' used for calculating the derivatives dzc_dxs,dzc_dys

	l_2 *= l_2;

	// step 1: calculate the EWA ellipse coefficients

	// compute normalized viewing vector V
	// restore camera coordinates of projected point (on z=1 plane)
	V_x = -(x0 / vp_sx - vp_tx);
	V_y = -(y0 / vp_sy - vp_ty);
	V_z = -1.f;
	r_ = 1.f / (float)sqrt(V_x * V_x + V_y * V_y + 1.f);
	V_x *= r_; V_y *= r_; V_z *= r_;
	ndotv = n[0] * V_x + n[1] * V_y + n[2] * V_z;

	if (ndotv < 0) {
		n[0] = -n[0];
		n[1] = -n[1];
		n[2] = -n[2];
		ndotv = -ndotv;
	}

	// project screen x,y unit vectors along V onto ST plane
	// determine direction Ix,Iy of projection of x,y
	Ix_x = -V_z * n[2] - V_y * n[1];		// (X x V) x N
	Ix_y = V_y * n[0];
	Ix_z = V_z * n[0];
	Iy_x = V_x * n[1];				// (Y x V) x N
	Iy_y = -V_x * n[0] - V_z * n[2];
	Iy_z = V_z * n[1];

	// f given by the vector equation Y+g*V=f*Iy resp X+g*V=f*Ix
	// Iy*f resp. Ix*f is the intersection point of Iy,Ix with Y+g*V resp. X+g*V
	// (see SurfaceSplatting.mws)
	f = -1.f / ndotv;

	// dzdx, dzdy derivatives for rasterizing z values on the ellipse
	if (f < -threshold_c3) e = -threshold_c3; else e = f;
	// note: since z coordinates in the z buffer are camera coordinate z-values, we have
	// to transform from screen to camera coordinates (by concatenating the screen-to-object
	// and object-to-camera scalings).
	dzc_dxs = Ix_z * e * scale_stoo * scale_otoc * z;
	dzc_dys = Iy_z * e * scale_stoo * scale_otoc * z;
	e = -e;

	// normalize Ix
	r = (float)sqrt(Ix_x * Ix_x + Ix_y * Ix_y + Ix_z * Ix_z);
	r_ = 1 / r;
	S_x = Ix_x * r_;
	S_y = Ix_y * r_;
	S_z = Ix_z * r_;

	// determine T parameter direction on ST plane
	// note: S parameter direction is chosen as Ix
	// T is automatically normalized,  since n and S have unit length
	// and are orthogonal by construction of S
	T_x = n[1] * S_z - n[2] * S_y;		// N x S
	T_y = n[2] * S_x - n[0] * S_z;
	T_z = n[0] * S_y - n[1] * S_x;

	// compute sx, sy, tx, ty derivatives
	// these derivatives build the inverse jacobian inv(J)=[sx,sy | tx,ty]
	// of the mapping J from object surface to screen

	_radius = surfel->radius;
	scale_otoc_radius = scale_otoc * _radius;
	_radius = 1.0f / _radius;

	Iy_x *= f; Iy_y *= f; Iy_z *= f;		// f*Iy is the intersection point with Y+g*V
	sx = r * f * scale_stoo * z * _radius * _l_2;		// note f given by the vector equation X+g*V=f*Ix takes the same value as above
	tx = 0;
	sy = (Iy_x * S_x + Iy_y * S_y + Iy_z * S_z) * scale_stoo * z * _radius * _l_2;		// Iy*S projects Iy onto S
	ty = (Iy_x * T_x + Iy_y * T_y + Iy_z * T_z) * scale_stoo * z * _radius * _l_2;		// Iy*T projects Iy onto T

	// compute z-range of the reconstruction kernel
	// see ellipseboundingbox_general.mws, idea: compute the point on the ellipse, where the ellipse tangent
	// is perpendicular to the depth gradient, i.e. [dzc_dxs, dzc_dys]. this is the point with maximum depth
	// on the ellipse
	// NOTE: to avoid overshoots in z, we use the z-range of the reconstruction kernel for blending!
	// NOTE: the variable "d" in the maple sheet corresponds to "-zbf_cutoffRadius_2"!
	// first, compute the conic matrix of the reconstruction kernel, which is [a b/2 | b/2 c] = inv(J)^T*inv(J)
	a = sx * sx;
	b_2 = sx * sy;
	b = 2.f * b_2;
	c = sy * sy + ty * ty;
	float discr;
	discr = -4 * a * dzc_dxs * b * dzc_dys * c - a * dzc_dys * dzc_dys * b * b + 4 * c * dzc_dys * dzc_dys * a * a +
		4 * a * dzc_dxs * dzc_dxs * c * c + b * b * b * dzc_dxs * dzc_dys - b * b * dzc_dxs * dzc_dxs * c;

	float zExtremum_x, zExtremum_y, tmp;
	tmp = (float)sqrt(discr * zbf_cutoffRadius_2);
	zExtremum_x = tmp * (-dzc_dys * b + 2 * dzc_dxs * c) / discr;
	zExtremum_y = tmp * (-2 * dzc_dys * a + dzc_dxs * b) / discr;

	float zRange_x, zRange_y;
	tmp = zExtremum_x * dzc_dxs;
	zRange_x = (tmp < 0) ? -tmp : tmp;
	tmp = zExtremum_y * dzc_dys;
	zRange_y = (tmp < 0) ? -tmp : tmp;

	float zMax, zMin;
	zMax = z + zRange_x + zRange_y;
	zMin = z - zRange_x - zRange_y;

	// guarantee a certain minimum z-range, otherwise blending fails for splats exactly parallel to the
	// image plane (the minimum z-range is 1 in object space, and then scaled to camera space).
	if (zMax - zMin < l_2 * scale_otoc_radius) {
		zMax += 0.5f * l_2 * scale_otoc_radius;
		zMin -= 0.5f * l_2 * scale_otoc_radius;
	}

	// calculate the matrix e[]=inv(J*transpose(J)+I), which describes the
	// EWA ellipse on the screen
	// note: column vectors are used, therefore the calculations are slightly
	// different to Heckbert's notation (cf. Maple file SurfaceSplatting2.mws)

	// the filtered variance matrix m[]
	// m[] = J*transpose(J)+I (details see SurfaceSplatting2.mws)
	r_ = 1.f / (sx * ty);
	r_ *= r_;							// r_ = 1/(sx*ty)^2, note that always tx=0, therefore some terms are missing
	m11 = (ty * ty + sy * sy) * r_ + 1.0f;	// in these formulas compared to the maple sheet
	m12 = -sy * sx * r_;
	m22 = sx * sx * r_ + 1.0f;

	// the filtered conic matric m[]^(-1)
	// matrix(A,B/2,B/2,C) = inv(m[])
	r_ = 1.f / (m11 * m22 - m12 * m12);
	a = m22 * r_;
	b_2 = -m12 * r_;
	b = b_2 * 2.f;
	c = m11 * r_;

	// calculate the normalization factor for the gaussian filter, which corresponds to the 
	// area of the reconstruction filter in source space, thus it is 
	// 1 / sqrt( det(inv(J)*transpose(inv(J))+I) )
	// note: the factor l_2 compensate for the level of detail of the LDC tree
	det_ = l_2 / (float)sqrt(sx * sx * ty * ty + sx * sx + sy * sy + ty * ty + 1.f);

	// bounding box of the ellipse
	// see ellipseboundingbox.mws, an exact axis aligned bounding box is computed by finding the points on
	// the ellipse where the tangent of the ellipse is parallel to x- and y-axis respectively.
	// NOTE: the variable "d" in the maple sheet corresponds to "-zbf_cutoffRadius_2"!
	discr = (float)sqrt((-b * b + 4 * c * a) * zbf_cutoffRadius_2 * a);
	ly = 2.f / (-b * b + 4 * c * a) * discr;

	discr = (float)sqrt(c * (-b * b + 4 * c * a) * zbf_cutoffRadius_2);
	lx = 2.f / (-b * b + 4 * c * a) * discr;

	lx = (lx < 0) ? -lx : lx;
	ly = (ly < 0) ? -ly : ly;
	Xmax = (int)(x0 + lx) + 1;
	Xmin = (int)(x0 - lx);
	Ymax = (int)(y0 + ly) + 1;
	Ymin = (int)(y0 - ly);

	// get surfel color
	r_comp = surfel->r;
	g_comp = surfel->g;
	b_comp = surfel->b;

	// step 2: rasterize the EWA ellipse

	// padding
	if (Xmin < 0) {
		Xmin = 0;
		if (Xmax < 0)
			return;
	}
	if (Xmax >= width) {
		Xmax = width - 1;
		if (Xmin >= width)
			return;
	}
	if (Ymin < 0) {
		Ymin = 0;
		if (Ymax < 0)
			return;
	}
	if (Ymax >= height) {
		Ymax = height - 1;
		if (Ymin >= height)
			return;
	}

	// set bounding box (the bounding box is a return value)
	bbox[0] = Xmin;
	bbox[1] = Ymin;
	bbox[2] = Xmax;
	bbox[3] = Ymax;

	// z value in the lower left corner of the rasterized area
	z_start = z - dzc_dxs * (x0 - ((float)Xmin + 0.5f)) - dzc_dys * (y0 - ((float)Ymin + 0.5f));

	x = ((float)Xmin + 0.5f) - x0;
	ddq = 2 * a;

	// *********************
	// ellipse rasterization
	// *********************
	for (Y = Ymin; Y <= Ymax; Y++)
	{
		// finite differences for ellipse rasterization
		y = ((float)Y + 0.5f) - y0;
		dq = a * (2 * x + 1) + b * y;
		q = (c * y + b * x) * y + a * x * x;

		// init z value
		z_cur = z_start;

		for (X = Xmin; X <= Xmax; X++)
		{
			i = X + width * Y;

			if (q < zbf_cutoffRadius_2)
			{
				// compare z-ranges
				if (zMin <= zBuffer[i].zMax)
				{
					// merge contributions
					w = filterLUT[(int)(q * _zbf_cutoffRadius_2 * zbf_LUTsize)] * det_;

					zBuffer[i].w += w;

					// add color contribution
					zBuffer[i].c[0] += r_comp * w;
					zBuffer[i].c[1] += g_comp * w;
					zBuffer[i].c[2] += b_comp * w;

					// normals
					zBuffer[i].n[0] += n[0] * w;
					zBuffer[i].n[1] += n[1] * w;
					zBuffer[i].n[2] += n[2] * w;
				}
			}	
			q += dq;
			dq += ddq;

			z_cur += dzc_dxs;
		}
		z_start += dzc_dys;
	}
}

void projectSampleStep1(int width, int height, Warper* warper, ZBufferProperty* zBufferProperty, ZBufferItem* zBuffer, float* filterLUT, Surfel* surfel, int bbox[4])
{
	int i, j;

	float wrp_frustum_nearplane;
	float wrp_frustum_farplane;
	float vp_sx, vp_sy;				// scaling for viewport mapping
	float vp_tx[3];							// translation for viewport mapping, x direction
	float vp_ty[3];							// translation for viewport mapping, y direction
	float A[3][9];					// the transformation matrix (in 3 variants, one for each base plane)
	float v[3];						// the translation vector
	float normalsA[9];				// the transposed inverse of A for transforming normals from camera to object space
	float stoo_scaling;				// screen to camera scaling due to viewport mapping and uniform scaling in the modelview transform
	float otoc_scaling;				// object to camera space scaling, due to scaling in transformation matrix

	// static variables used for warping, which are independent of current block
	wrp_frustum_nearplane = warper->frustum.nearplane;
	wrp_frustum_farplane = warper->frustum.farplane;
	stoo_scaling = warper->frustum.xP * 2 / (width * warper->transformation.scaling);
	otoc_scaling = warper->transformation.scaling;

	// set transformation variables
	memcpy(A[0], warper->transformation.rotation, sizeof(float) * 9);
	//MtrCopy3x3f(warper->transformation.rotation, A[0]);
	for (i = 0; i < 3; i++) v[i] = warper->transformation.translation[i];
	memcpy(normalsA, warper->transformation.normalsRotation, sizeof(float) * 9);
	//MtrCopy3x3f(warper->transformation.normalsRotation, normalsA);

	// rotation matrices for yz- and zx-baseplanes
	for (i = 0; i < 3; i++)
	{
		for (j = 0; j < 3; j++)
		{
			A[1][i * 3 + j] = A[0][i * 3 + (j + 1) % 3];
			A[2][i * 3 + j] = A[0][i * 3 + (j + 2) % 3];
		}
	}

	// set viewport mapping variables
	vp_sx = width / (2 * warper->frustum.xP);
	vp_sy = height / (2 * warper->frustum.yP);

	// set variables for warping from all 3 baseplanes
	for (i = 0; i < 3; i++)
	{
		vp_tx[i] = warper->frustum.xP - warper->frustum.xC;
		vp_ty[i] = warper->frustum.yP - warper->frustum.yC;
	}

	float x_c, y_c;             // camera-space x and y values
	float z_c, r_z_c;			// camera-space z-value (and its reciprocal) of sample being warped

	float xImg, yImg;			// x- and y-screen-coordinates of warped sample
	float xPad, yPad;			// pads in x and y direction for correct clipping

	Vector3D pos;				// object space sample position
	Vector3D nrm;				// object space normal
	Vector3D tan1,              // object space tangent axes
		tan2;
	float r;					// surfel radius
	float n[3];					// camera space normal
	int splatBBox[4];			// bounding box of rasterized splat

	// get sample position
	pos = surfel->position;

	// apply transformation matrix
	z_c = A[0][6] * pos[0] + A[0][7] * pos[1] + A[0][8] * pos[2] + v[2];
	// apply near and far clipping planes
	if (z_c > wrp_frustum_nearplane && z_c < wrp_frustum_farplane) {

		x_c = A[0][0] * pos[0] + A[0][1] * pos[1] + A[0][2] * pos[2] + v[0];
		y_c = A[0][3] * pos[0] + A[0][4] * pos[1] + A[0][5] * pos[2] + v[1];

		// perspective divide and viewport transformation
		r_z_c = 1 / z_c;
		xImg = (x_c * r_z_c + vp_tx[0]) * vp_sx;
		yImg = (y_c * r_z_c + vp_ty[0]) * vp_sy;

		// for correct clipping: project surfel radius to screen space
		r = surfel->radius;
		r *= 1 / z_c;
		xPad = r * vp_sx;
		yPad = r * vp_sy;

		// put it into the z-buffer
		if ((xImg >= -xPad) && (xImg < width + xPad) &&
			(yImg >= -yPad) && (yImg < height + yPad))
		{
			// transform normal to camera coordinates
			nrm = surfel->normal;

			n[0] = normalsA[0] * nrm[0] + normalsA[1] * nrm[1] + normalsA[2] * nrm[2];
			n[1] = normalsA[3] * nrm[0] + normalsA[4] * nrm[1] + normalsA[5] * nrm[2];
			n[2] = normalsA[6] * nrm[0] + normalsA[7] * nrm[1] + normalsA[8] * nrm[2];

			// caution: this function (or macro) relies on global variables!
			// note: 'warped level' is set to 0
			surfaceSplatStep1(width, height, zBufferProperty, zBuffer, filterLUT, xImg, yImg, z_c, n, surfel, 0, stoo_scaling, otoc_scaling, vp_sx, vp_sy, vp_tx[0], vp_ty[0], splatBBox);

			// enlarge bounding box (containing the modified frame buffer area) if necessary
			if (splatBBox[0] < bbox[0]) bbox[0] = splatBBox[0];
			if (splatBBox[1] < bbox[1]) bbox[1] = splatBBox[1];
			if (splatBBox[2] > bbox[2]) bbox[2] = splatBBox[2];
			if (splatBBox[3] > bbox[3]) bbox[3] = splatBBox[3];
		}
	}
}

void projectSampleStep2(int width, int height, Warper* warper, ZBufferProperty* zBufferProperty, ZBufferItem* zBuffer, float* filterLUT, Surfel* surfel, int bbox[4])
{
	int i, j;

	float wrp_frustum_nearplane;
	float wrp_frustum_farplane;
	float vp_sx, vp_sy;				// scaling for viewport mapping
	float vp_tx[3];							// translation for viewport mapping, x direction
	float vp_ty[3];							// translation for viewport mapping, y direction
	float A[3][9];					// the transformation matrix (in 3 variants, one for each base plane)
	float v[3];						// the translation vector
	float normalsA[9];				// the transposed inverse of A for transforming normals from camera to object space
	float stoo_scaling;				// screen to camera scaling due to viewport mapping and uniform scaling in the modelview transform
	float otoc_scaling;				// object to camera space scaling, due to scaling in transformation matrix

	// static variables used for warping, which are independent of current block
	wrp_frustum_nearplane = warper->frustum.nearplane;
	wrp_frustum_farplane = warper->frustum.farplane;
	stoo_scaling = warper->frustum.xP * 2 / (width * warper->transformation.scaling);
	otoc_scaling = warper->transformation.scaling;

	// set transformation variables
	memcpy(A[0], warper->transformation.rotation, sizeof(float) * 9);
	//MtrCopy3x3f(warper->transformation.rotation, A[0]);
	for (i = 0; i < 3; i++) v[i] = warper->transformation.translation[i];
	memcpy(normalsA, warper->transformation.normalsRotation, sizeof(float) * 9);
	//MtrCopy3x3f(warper->transformation.normalsRotation, normalsA);

	// rotation matrices for yz- and zx-baseplanes
	for (i = 0; i < 3; i++)
	{
		for (j = 0; j < 3; j++)
		{
			A[1][i * 3 + j] = A[0][i * 3 + (j + 1) % 3];
			A[2][i * 3 + j] = A[0][i * 3 + (j + 2) % 3];
		}
	}

	// set viewport mapping variables
	vp_sx = width / (2 * warper->frustum.xP);
	vp_sy = height / (2 * warper->frustum.yP);

	// set variables for warping from all 3 baseplanes
	for (i = 0; i < 3; i++)
	{
		vp_tx[i] = warper->frustum.xP - warper->frustum.xC;
		vp_ty[i] = warper->frustum.yP - warper->frustum.yC;
	}

	float x_c, y_c;             // camera-space x and y values
	float z_c, r_z_c;			// camera-space z-value (and its reciprocal) of sample being warped

	float xImg, yImg;			// x- and y-screen-coordinates of warped sample
	float xPad, yPad;			// pads in x and y direction for correct clipping

	Vector3D pos;				// object space sample position
	Vector3D nrm;				// object space normal
	Vector3D tan1,              // object space tangent axes
		tan2;
	float r;					// surfel radius
	float n[3];					// camera space normal
	int splatBBox[4];			// bounding box of rasterized splat

	// get sample position
	pos = surfel->position;

	// apply transformation matrix
	z_c = A[0][6] * pos[0] + A[0][7] * pos[1] + A[0][8] * pos[2] + v[2];
	// apply near and far clipping planes
	if (z_c > wrp_frustum_nearplane && z_c < wrp_frustum_farplane) {

		x_c = A[0][0] * pos[0] + A[0][1] * pos[1] + A[0][2] * pos[2] + v[0];
		y_c = A[0][3] * pos[0] + A[0][4] * pos[1] + A[0][5] * pos[2] + v[1];

		// perspective divide and viewport transformation
		r_z_c = 1 / z_c;
		xImg = (x_c * r_z_c + vp_tx[0]) * vp_sx;
		yImg = (y_c * r_z_c + vp_ty[0]) * vp_sy;

		// for correct clipping: project surfel radius to screen space
		r = surfel->radius;
		r *= 1 / z_c;
		xPad = r * vp_sx;
		yPad = r * vp_sy;

		// put it into the z-buffer
		if ((xImg >= -xPad) && (xImg < width + xPad) &&
			(yImg >= -yPad) && (yImg < height + yPad))
		{
			// transform normal to camera coordinates
			nrm = surfel->normal;

			n[0] = normalsA[0] * nrm[0] + normalsA[1] * nrm[1] + normalsA[2] * nrm[2];
			n[1] = normalsA[3] * nrm[0] + normalsA[4] * nrm[1] + normalsA[5] * nrm[2];
			n[2] = normalsA[6] * nrm[0] + normalsA[7] * nrm[1] + normalsA[8] * nrm[2];

			// caution: this function (or macro) relies on global variables!
			// note: 'warped level' is set to 0
			surfaceSplatStep2(width, height, zBufferProperty, zBuffer, filterLUT, xImg, yImg, z_c, n, surfel, 0, stoo_scaling, otoc_scaling, vp_sx, vp_sy, vp_tx[0], vp_ty[0], splatBBox);

			// enlarge bounding box (containing the modified frame buffer area) if necessary
			if (splatBBox[0] < bbox[0]) bbox[0] = splatBBox[0];
			if (splatBBox[1] < bbox[1]) bbox[1] = splatBBox[1];
			if (splatBBox[2] > bbox[2]) bbox[2] = splatBBox[2];
			if (splatBBox[3] > bbox[3]) bbox[3] = splatBBox[3];
		}
	}
}

void project(int width, int height, Warper* warper, ZBufferProperty* zBufferProperty, ZBufferItem* zBuffer, float* filterLUT, int numSurfels, Surfel* surfels, int bbox[4]) {
	for (int i = 0; i < numSurfels; i++)
		projectSampleStep1(width, height, warper, zBufferProperty, zBuffer, filterLUT, &surfels[i], bbox);
	for (int i = 0; i < numSurfels; i++)
		projectSampleStep2(width, height, warper, zBufferProperty, zBuffer, filterLUT, &surfels[i], bbox);
}

__device__ static float atomicMin(float* address, float val)
{
	int* address_as_i = (int*)address;
	int old = *address_as_i, assumed;
	do {
		assumed = old;
		old = ::atomicCAS(address_as_i, assumed,
			__float_as_int(::fminf(val, __int_as_float(assumed))));
	} while (assumed != old);
	return __int_as_float(old);
}

__device__ static float atomicMax(float* address, float val)
{
	int* address_as_i = (int*)address;
	int old = *address_as_i, assumed;
	do {
		assumed = old;
		old = ::atomicCAS(address_as_i, assumed,
			__float_as_int(::fmaxf(val, __int_as_float(assumed))));
	} while (assumed != old);
	return __int_as_float(old);
}

__device__ void surfaceSplatStep1GPU(int width, int height, ZBufferProperty* zBufferProperty, ZBufferItem* zBuffer, float* filterLUT, float x0, float y0, float z, float n[3], Surfel* surfel, int l, float scale_stoo, float scale_otoc, float vp_sx, float vp_sy, float vp_tx, float vp_ty, int* bbox)
{
	float zbf_LUTsize = zBufferProperty->LUTsize;
	float zbf_cutoffRadius_2 = zBufferProperty->cutoffRadius * zBufferProperty->cutoffRadius;
	float _zbf_cutoffRadius_2 = 1 / zbf_cutoffRadius_2;
	float zbf_angleThreshold = zBufferProperty->angleTrheshold;

	float V_x, V_y, V_z;		// viewing direction
	float S_x, S_y, S_z;		// S parameter direction on ST plane
	float T_x, T_y, T_z;		// T parameter direction on ST plane
	float Ix_x, Ix_y, Ix_z;		// direction of projection of screen x vector onto ST plane
	float Iy_x, Iy_y, Iy_z;		// direction of projection of screen y vector onto ST plane
	float r, r_, f;
	float ndotv;				// N*V (normal dot viewing direction) dotproduct
	float sx, sy, tx, ty;		// derivatives of the screen to ST plane mapping
	float m11, m12, m22;
	float a, b, b_2, c;			// the EWA ellipse coefficients

	int Xmin, Xmax, Ymin, Ymax;	// bounding box of the ellipse to be rasterized
	float lx, ly;
	int X, Y;
	float x, y;
	float q, dq, ddq;

	float dzc_dxs, dzc_dys;			// dzc/dxs, dzc/dys derivatives
	float z_start, z_cur;			// z values to be rasterized

	int i;
	float r_comp, g_comp, b_comp;

	float threshold_c3;
	float e;
	float w;

	float l_2, _l_2;
	float det_;

	Vector3D normal;
	float _radius;

	//scale z buffer according to surfel radius
	float scale_otoc_radius;

	// init bounding box return value
	bbox[0] = width;
	bbox[1] = height;
	bbox[2] = bbox[3] = 0;

	l_2 = (float)(1 << l);
	_l_2 = 1 / l_2;

	// thresholds for the 'merge/separate' decision
	// note: 
	// - these thresholds are dependent on the level of the warped samples, since the
	// distance on the surface between samples increases with the warped level.
	// - these offsets should be constant in units 'dz per unit in object space'. but since 
	// z coordinates in the z-buffer are in camera coordinates, we have to transform the offsets
	// to camera space by applying the 'scale_otoc' (object to camera space) scaling
	threshold_c3 = zbf_angleThreshold;							// cut off for 'ndotv' used for calculating the derivatives dzc_dxs,dzc_dys

	l_2 *= l_2;

	// step 1: calculate the EWA ellipse coefficients

	// compute normalized viewing vector V
	// restore camera coordinates of projected point (on z=1 plane)
	V_x = -(x0 / vp_sx - vp_tx);
	V_y = -(y0 / vp_sy - vp_ty);
	V_z = -1.f;
	r_ = 1.f / (float)sqrt(V_x * V_x + V_y * V_y + 1.f);
	V_x *= r_; V_y *= r_; V_z *= r_;
	ndotv = n[0] * V_x + n[1] * V_y + n[2] * V_z;

	if (ndotv < 0) {
		n[0] = -n[0];
		n[1] = -n[1];
		n[2] = -n[2];
		ndotv = -ndotv;
	}

	// project screen x,y unit vectors along V onto ST plane
	// determine direction Ix,Iy of projection of x,y
	Ix_x = -V_z * n[2] - V_y * n[1];		// (X x V) x N
	Ix_y = V_y * n[0];
	Ix_z = V_z * n[0];
	Iy_x = V_x * n[1];				// (Y x V) x N
	Iy_y = -V_x * n[0] - V_z * n[2];
	Iy_z = V_z * n[1];

	// f given by the vector equation Y+g*V=f*Iy resp X+g*V=f*Ix
	// Iy*f resp. Ix*f is the intersection point of Iy,Ix with Y+g*V resp. X+g*V
	// (see SurfaceSplatting.mws)
	f = -1.f / ndotv;

	// dzdx, dzdy derivatives for rasterizing z values on the ellipse
	if (f < -threshold_c3) e = -threshold_c3; else e = f;
	// note: since z coordinates in the z buffer are camera coordinate z-values, we have
	// to transform from screen to camera coordinates (by concatenating the screen-to-object
	// and object-to-camera scalings).
	dzc_dxs = Ix_z * e * scale_stoo * scale_otoc * z;
	dzc_dys = Iy_z * e * scale_stoo * scale_otoc * z;
	e = -e;

	// normalize Ix
	r = (float)sqrt(Ix_x * Ix_x + Ix_y * Ix_y + Ix_z * Ix_z);
	r_ = 1 / r;
	S_x = Ix_x * r_;
	S_y = Ix_y * r_;
	S_z = Ix_z * r_;

	// determine T parameter direction on ST plane
	// note: S parameter direction is chosen as Ix
	// T is automatically normalized,  since n and S have unit length
	// and are orthogonal by construction of S
	T_x = n[1] * S_z - n[2] * S_y;		// N x S
	T_y = n[2] * S_x - n[0] * S_z;
	T_z = n[0] * S_y - n[1] * S_x;

	// compute sx, sy, tx, ty derivatives
	// these derivatives build the inverse jacobian inv(J)=[sx,sy | tx,ty]
	// of the mapping J from object surface to screen

	_radius = surfel->radius;
	scale_otoc_radius = scale_otoc * _radius;
	_radius = 1.0f / _radius;

	Iy_x *= f; Iy_y *= f; Iy_z *= f;		// f*Iy is the intersection point with Y+g*V
	sx = r * f * scale_stoo * z * _radius * _l_2;		// note f given by the vector equation X+g*V=f*Ix takes the same value as above
	tx = 0;
	sy = (Iy_x * S_x + Iy_y * S_y + Iy_z * S_z) * scale_stoo * z * _radius * _l_2;		// Iy*S projects Iy onto S
	ty = (Iy_x * T_x + Iy_y * T_y + Iy_z * T_z) * scale_stoo * z * _radius * _l_2;		// Iy*T projects Iy onto T

	// compute z-range of the reconstruction kernel
	// see ellipseboundingbox_general.mws, idea: compute the point on the ellipse, where the ellipse tangent
	// is perpendicular to the depth gradient, i.e. [dzc_dxs, dzc_dys]. this is the point with maximum depth
	// on the ellipse
	// NOTE: to avoid overshoots in z, we use the z-range of the reconstruction kernel for blending!
	// NOTE: the variable "d" in the maple sheet corresponds to "-zbf_cutoffRadius_2"!
	// first, compute the conic matrix of the reconstruction kernel, which is [a b/2 | b/2 c] = inv(J)^T*inv(J)
	a = sx * sx;
	b_2 = sx * sy;
	b = 2.f * b_2;
	c = sy * sy + ty * ty;
	float discr;
	discr = -4 * a * dzc_dxs * b * dzc_dys * c - a * dzc_dys * dzc_dys * b * b + 4 * c * dzc_dys * dzc_dys * a * a +
		4 * a * dzc_dxs * dzc_dxs * c * c + b * b * b * dzc_dxs * dzc_dys - b * b * dzc_dxs * dzc_dxs * c;

	float zExtremum_x, zExtremum_y, tmp;
	tmp = (float)sqrt(discr * zbf_cutoffRadius_2);
	zExtremum_x = tmp * (-dzc_dys * b + 2 * dzc_dxs * c) / discr;
	zExtremum_y = tmp * (-2 * dzc_dys * a + dzc_dxs * b) / discr;

	float zRange_x, zRange_y;
	tmp = zExtremum_x * dzc_dxs;
	zRange_x = (tmp < 0) ? -tmp : tmp;
	tmp = zExtremum_y * dzc_dys;
	zRange_y = (tmp < 0) ? -tmp : tmp;

	float zMax, zMin;
	zMax = z + zRange_x + zRange_y;
	zMin = z - zRange_x - zRange_y;

	// guarantee a certain minimum z-range, otherwise blending fails for splats exactly parallel to the
	// image plane (the minimum z-range is 1 in object space, and then scaled to camera space).
	if (zMax - zMin < l_2 * scale_otoc_radius) {
		zMax += 0.5f * l_2 * scale_otoc_radius;
		zMin -= 0.5f * l_2 * scale_otoc_radius;
	}

	// calculate the matrix e[]=inv(J*transpose(J)+I), which describes the
	// EWA ellipse on the screen
	// note: column vectors are used, therefore the calculations are slightly
	// different to Heckbert's notation (cf. Maple file SurfaceSplatting2.mws)

	// the filtered variance matrix m[]
	// m[] = J*transpose(J)+I (details see SurfaceSplatting2.mws)
	r_ = 1.f / (sx * ty);
	r_ *= r_;							// r_ = 1/(sx*ty)^2, note that always tx=0, therefore some terms are missing
	m11 = (ty * ty + sy * sy) * r_ + 1.0f;	// in these formulas compared to the maple sheet
	m12 = -sy * sx * r_;
	m22 = sx * sx * r_ + 1.0f;

	// the filtered conic matric m[]^(-1)
	// matrix(A,B/2,B/2,C) = inv(m[])
	r_ = 1.f / (m11 * m22 - m12 * m12);
	a = m22 * r_;
	b_2 = -m12 * r_;
	b = b_2 * 2.f;
	c = m11 * r_;

	// calculate the normalization factor for the gaussian filter, which corresponds to the 
	// area of the reconstruction filter in source space, thus it is 
	// 1 / sqrt( det(inv(J)*transpose(inv(J))+I) )
	// note: the factor l_2 compensate for the level of detail of the LDC tree
	det_ = l_2 / (float)sqrt(sx * sx * ty * ty + sx * sx + sy * sy + ty * ty + 1.f);

	// bounding box of the ellipse
	// see ellipseboundingbox.mws, an exact axis aligned bounding box is computed by finding the points on
	// the ellipse where the tangent of the ellipse is parallel to x- and y-axis respectively.
	// NOTE: the variable "d" in the maple sheet corresponds to "-zbf_cutoffRadius_2"!
	discr = (float)sqrt((-b * b + 4 * c * a) * zbf_cutoffRadius_2 * a);
	ly = 2.f / (-b * b + 4 * c * a) * discr;

	discr = (float)sqrt(c * (-b * b + 4 * c * a) * zbf_cutoffRadius_2);
	lx = 2.f / (-b * b + 4 * c * a) * discr;

	lx = (lx < 0) ? -lx : lx;
	ly = (ly < 0) ? -ly : ly;
	Xmax = (int)(x0 + lx) + 1;
	Xmin = (int)(x0 - lx);
	Ymax = (int)(y0 + ly) + 1;
	Ymin = (int)(y0 - ly);

	// get surfel color
	r_comp = surfel->r;
	g_comp = surfel->g;
	b_comp = surfel->b;

	// step 2: rasterize the EWA ellipse

	// padding
	if (Xmin < 0) {
		Xmin = 0;
		if (Xmax < 0)
			return;
	}
	if (Xmax >= width) {
		Xmax = width - 1;
		if (Xmin >= width)
			return;
	}
	if (Ymin < 0) {
		Ymin = 0;
		if (Ymax < 0)
			return;
	}
	if (Ymax >= height) {
		Ymax = height - 1;
		if (Ymin >= height)
			return;
	}

	// set bounding box (the bounding box is a return value)
	bbox[0] = Xmin;
	bbox[1] = Ymin;
	bbox[2] = Xmax;
	bbox[3] = Ymax;

	// z value in the lower left corner of the rasterized area
	z_start = z - dzc_dxs * (x0 - ((float)Xmin + 0.5f)) - dzc_dys * (y0 - ((float)Ymin + 0.5f));

	x = ((float)Xmin + 0.5f) - x0;
	ddq = 2 * a;

	// *********************
	// ellipse rasterization
	// *********************
	for (Y = Ymin; Y <= Ymax; Y++)
	{
		// finite differences for ellipse rasterization
		y = ((float)Y + 0.5f) - y0;
		dq = a * (2 * x + 1) + b * y;
		q = (c * y + b * x) * y + a * x * x;

		// init z value
		z_cur = z_start;

		for (X = Xmin; X <= Xmax; X++)
		{
			i = X + width * Y;

			if (q < zbf_cutoffRadius_2)
				atomicMin(&zBuffer[i].zMin, zMin);
			q += dq;
			dq += ddq;

			z_cur += dzc_dxs;
		}
		z_start += dzc_dys;
	}
}

__device__ void surfaceSplatStep2GPU(int width, int height, ZBufferProperty* zBufferProperty, ZBufferItem* zBuffer, float* filterLUT, float x0, float y0, float z, float n[3], Surfel* surfel, int l, float scale_stoo, float scale_otoc, float vp_sx, float vp_sy, float vp_tx, float vp_ty, int* bbox)
{
	float zbf_LUTsize = zBufferProperty->LUTsize;
	float zbf_cutoffRadius_2 = zBufferProperty->cutoffRadius * zBufferProperty->cutoffRadius;
	float _zbf_cutoffRadius_2 = 1 / zbf_cutoffRadius_2;
	float zbf_angleThreshold = zBufferProperty->angleTrheshold;

	float V_x, V_y, V_z;		// viewing direction
	float S_x, S_y, S_z;		// S parameter direction on ST plane
	float T_x, T_y, T_z;		// T parameter direction on ST plane
	float Ix_x, Ix_y, Ix_z;		// direction of projection of screen x vector onto ST plane
	float Iy_x, Iy_y, Iy_z;		// direction of projection of screen y vector onto ST plane
	float r, r_, f;
	float ndotv;				// N*V (normal dot viewing direction) dotproduct
	float sx, sy, tx, ty;		// derivatives of the screen to ST plane mapping
	float m11, m12, m22;
	float a, b, b_2, c;			// the EWA ellipse coefficients

	int Xmin, Xmax, Ymin, Ymax;	// bounding box of the ellipse to be rasterized
	float lx, ly;
	int X, Y;
	float x, y;
	float q, dq, ddq;

	float dzc_dxs, dzc_dys;			// dzc/dxs, dzc/dys derivatives
	float z_start, z_cur;			// z values to be rasterized

	int i;
	float r_comp, g_comp, b_comp;

	float threshold_c3;
	float e;
	float w;

	float l_2, _l_2;
	float det_;

	Vector3D normal;
	float _radius;

	//scale z buffer according to surfel radius
	float scale_otoc_radius;

	// init bounding box return value
	bbox[0] = width;
	bbox[1] = height;
	bbox[2] = bbox[3] = 0;

	l_2 = (float)(1 << l);
	_l_2 = 1 / l_2;

	// thresholds for the 'merge/separate' decision
	// note: 
	// - these thresholds are dependent on the level of the warped samples, since the
	// distance on the surface between samples increases with the warped level.
	// - these offsets should be constant in units 'dz per unit in object space'. but since 
	// z coordinates in the z-buffer are in camera coordinates, we have to transform the offsets
	// to camera space by applying the 'scale_otoc' (object to camera space) scaling
	threshold_c3 = zbf_angleThreshold;							// cut off for 'ndotv' used for calculating the derivatives dzc_dxs,dzc_dys

	l_2 *= l_2;

	// step 1: calculate the EWA ellipse coefficients

	// compute normalized viewing vector V
	// restore camera coordinates of projected point (on z=1 plane)
	V_x = -(x0 / vp_sx - vp_tx);
	V_y = -(y0 / vp_sy - vp_ty);
	V_z = -1.f;
	r_ = 1.f / (float)sqrt(V_x * V_x + V_y * V_y + 1.f);
	V_x *= r_; V_y *= r_; V_z *= r_;
	ndotv = n[0] * V_x + n[1] * V_y + n[2] * V_z;

	if (ndotv < 0) {
		n[0] = -n[0];
		n[1] = -n[1];
		n[2] = -n[2];
		ndotv = -ndotv;
	}

	// project screen x,y unit vectors along V onto ST plane
	// determine direction Ix,Iy of projection of x,y
	Ix_x = -V_z * n[2] - V_y * n[1];		// (X x V) x N
	Ix_y = V_y * n[0];
	Ix_z = V_z * n[0];
	Iy_x = V_x * n[1];				// (Y x V) x N
	Iy_y = -V_x * n[0] - V_z * n[2];
	Iy_z = V_z * n[1];

	// f given by the vector equation Y+g*V=f*Iy resp X+g*V=f*Ix
	// Iy*f resp. Ix*f is the intersection point of Iy,Ix with Y+g*V resp. X+g*V
	// (see SurfaceSplatting.mws)
	f = -1.f / ndotv;

	// dzdx, dzdy derivatives for rasterizing z values on the ellipse
	if (f < -threshold_c3) e = -threshold_c3; else e = f;
	// note: since z coordinates in the z buffer are camera coordinate z-values, we have
	// to transform from screen to camera coordinates (by concatenating the screen-to-object
	// and object-to-camera scalings).
	dzc_dxs = Ix_z * e * scale_stoo * scale_otoc * z;
	dzc_dys = Iy_z * e * scale_stoo * scale_otoc * z;
	e = -e;

	// normalize Ix
	r = (float)sqrt(Ix_x * Ix_x + Ix_y * Ix_y + Ix_z * Ix_z);
	r_ = 1 / r;
	S_x = Ix_x * r_;
	S_y = Ix_y * r_;
	S_z = Ix_z * r_;

	// determine T parameter direction on ST plane
	// note: S parameter direction is chosen as Ix
	// T is automatically normalized,  since n and S have unit length
	// and are orthogonal by construction of S
	T_x = n[1] * S_z - n[2] * S_y;		// N x S
	T_y = n[2] * S_x - n[0] * S_z;
	T_z = n[0] * S_y - n[1] * S_x;

	// compute sx, sy, tx, ty derivatives
	// these derivatives build the inverse jacobian inv(J)=[sx,sy | tx,ty]
	// of the mapping J from object surface to screen

	_radius = surfel->radius;
	scale_otoc_radius = scale_otoc * _radius;
	_radius = 1.0f / _radius;

	Iy_x *= f; Iy_y *= f; Iy_z *= f;		// f*Iy is the intersection point with Y+g*V
	sx = r * f * scale_stoo * z * _radius * _l_2;		// note f given by the vector equation X+g*V=f*Ix takes the same value as above
	tx = 0;
	sy = (Iy_x * S_x + Iy_y * S_y + Iy_z * S_z) * scale_stoo * z * _radius * _l_2;		// Iy*S projects Iy onto S
	ty = (Iy_x * T_x + Iy_y * T_y + Iy_z * T_z) * scale_stoo * z * _radius * _l_2;		// Iy*T projects Iy onto T

	// compute z-range of the reconstruction kernel
	// see ellipseboundingbox_general.mws, idea: compute the point on the ellipse, where the ellipse tangent
	// is perpendicular to the depth gradient, i.e. [dzc_dxs, dzc_dys]. this is the point with maximum depth
	// on the ellipse
	// NOTE: to avoid overshoots in z, we use the z-range of the reconstruction kernel for blending!
	// NOTE: the variable "d" in the maple sheet corresponds to "-zbf_cutoffRadius_2"!
	// first, compute the conic matrix of the reconstruction kernel, which is [a b/2 | b/2 c] = inv(J)^T*inv(J)
	a = sx * sx;
	b_2 = sx * sy;
	b = 2.f * b_2;
	c = sy * sy + ty * ty;
	float discr;
	discr = -4 * a * dzc_dxs * b * dzc_dys * c - a * dzc_dys * dzc_dys * b * b + 4 * c * dzc_dys * dzc_dys * a * a +
		4 * a * dzc_dxs * dzc_dxs * c * c + b * b * b * dzc_dxs * dzc_dys - b * b * dzc_dxs * dzc_dxs * c;

	float zExtremum_x, zExtremum_y, tmp;
	tmp = (float)sqrt(discr * zbf_cutoffRadius_2);
	zExtremum_x = tmp * (-dzc_dys * b + 2 * dzc_dxs * c) / discr;
	zExtremum_y = tmp * (-2 * dzc_dys * a + dzc_dxs * b) / discr;

	float zRange_x, zRange_y;
	tmp = zExtremum_x * dzc_dxs;
	zRange_x = (tmp < 0) ? -tmp : tmp;
	tmp = zExtremum_y * dzc_dys;
	zRange_y = (tmp < 0) ? -tmp : tmp;

	float zMax, zMin;
	zMax = z + zRange_x + zRange_y;
	zMin = z - zRange_x - zRange_y;

	// guarantee a certain minimum z-range, otherwise blending fails for splats exactly parallel to the
	// image plane (the minimum z-range is 1 in object space, and then scaled to camera space).
	if (zMax - zMin < l_2 * scale_otoc_radius) {
		zMax += 0.5f * l_2 * scale_otoc_radius;
		zMin -= 0.5f * l_2 * scale_otoc_radius;
	}

	// calculate the matrix e[]=inv(J*transpose(J)+I), which describes the
	// EWA ellipse on the screen
	// note: column vectors are used, therefore the calculations are slightly
	// different to Heckbert's notation (cf. Maple file SurfaceSplatting2.mws)

	// the filtered variance matrix m[]
	// m[] = J*transpose(J)+I (details see SurfaceSplatting2.mws)
	r_ = 1.f / (sx * ty);
	r_ *= r_;							// r_ = 1/(sx*ty)^2, note that always tx=0, therefore some terms are missing
	m11 = (ty * ty + sy * sy) * r_ + 1.0f;	// in these formulas compared to the maple sheet
	m12 = -sy * sx * r_;
	m22 = sx * sx * r_ + 1.0f;

	// the filtered conic matric m[]^(-1)
	// matrix(A,B/2,B/2,C) = inv(m[])
	r_ = 1.f / (m11 * m22 - m12 * m12);
	a = m22 * r_;
	b_2 = -m12 * r_;
	b = b_2 * 2.f;
	c = m11 * r_;

	// calculate the normalization factor for the gaussian filter, which corresponds to the 
	// area of the reconstruction filter in source space, thus it is 
	// 1 / sqrt( det(inv(J)*transpose(inv(J))+I) )
	// note: the factor l_2 compensate for the level of detail of the LDC tree
	det_ = l_2 / (float)sqrt(sx * sx * ty * ty + sx * sx + sy * sy + ty * ty + 1.f);

	// bounding box of the ellipse
	// see ellipseboundingbox.mws, an exact axis aligned bounding box is computed by finding the points on
	// the ellipse where the tangent of the ellipse is parallel to x- and y-axis respectively.
	// NOTE: the variable "d" in the maple sheet corresponds to "-zbf_cutoffRadius_2"!
	discr = (float)sqrt((-b * b + 4 * c * a) * zbf_cutoffRadius_2 * a);
	ly = 2.f / (-b * b + 4 * c * a) * discr;

	discr = (float)sqrt(c * (-b * b + 4 * c * a) * zbf_cutoffRadius_2);
	lx = 2.f / (-b * b + 4 * c * a) * discr;

	lx = (lx < 0) ? -lx : lx;
	ly = (ly < 0) ? -ly : ly;
	Xmax = (int)(x0 + lx) + 1;
	Xmin = (int)(x0 - lx);
	Ymax = (int)(y0 + ly) + 1;
	Ymin = (int)(y0 - ly);

	// get surfel color
	r_comp = surfel->r;
	g_comp = surfel->g;
	b_comp = surfel->b;

	// step 2: rasterize the EWA ellipse

	// padding
	if (Xmin < 0) {
		Xmin = 0;
		if (Xmax < 0)
			return;
	}
	if (Xmax >= width) {
		Xmax = width - 1;
		if (Xmin >= width)
			return;
	}
	if (Ymin < 0) {
		Ymin = 0;
		if (Ymax < 0)
			return;
	}
	if (Ymax >= height) {
		Ymax = height - 1;
		if (Ymin >= height)
			return;
	}

	// set bounding box (the bounding box is a return value)
	bbox[0] = Xmin;
	bbox[1] = Ymin;
	bbox[2] = Xmax;
	bbox[3] = Ymax;

	// z value in the lower left corner of the rasterized area
	z_start = z - dzc_dxs * (x0 - ((float)Xmin + 0.5f)) - dzc_dys * (y0 - ((float)Ymin + 0.5f));

	x = ((float)Xmin + 0.5f) - x0;
	ddq = 2 * a;

	// *********************
	// ellipse rasterization
	// *********************
	for (Y = Ymin; Y <= Ymax; Y++)
	{
		// finite differences for ellipse rasterization
		y = ((float)Y + 0.5f) - y0;
		dq = a * (2 * x + 1) + b * y;
		q = (c * y + b * x) * y + a * x * x;

		// init z value
		z_cur = z_start;

		for (X = Xmin; X <= Xmax; X++)
		{
			i = X + width * Y;

			if (q < zbf_cutoffRadius_2)
				if (zMin == zBuffer[i].zMin)
					atomicMax(&zBuffer[i].zMax, zMax);
			q += dq;
			dq += ddq;

			z_cur += dzc_dxs;
		}
		z_start += dzc_dys;
	}
}

__device__ void surfaceSplatStep3GPU(int width, int height, ZBufferProperty* zBufferProperty, ZBufferItem* zBuffer, float* filterLUT, float x0, float y0, float z, float n[3], Surfel* surfel, int l, float scale_stoo, float scale_otoc, float vp_sx, float vp_sy, float vp_tx, float vp_ty, int* bbox)
{
	float zbf_LUTsize = zBufferProperty->LUTsize;
	float zbf_cutoffRadius_2 = zBufferProperty->cutoffRadius * zBufferProperty->cutoffRadius;
	float _zbf_cutoffRadius_2 = 1 / zbf_cutoffRadius_2;
	float zbf_angleThreshold = zBufferProperty->angleTrheshold;

	float V_x, V_y, V_z;		// viewing direction
	float S_x, S_y, S_z;		// S parameter direction on ST plane
	float T_x, T_y, T_z;		// T parameter direction on ST plane
	float Ix_x, Ix_y, Ix_z;		// direction of projection of screen x vector onto ST plane
	float Iy_x, Iy_y, Iy_z;		// direction of projection of screen y vector onto ST plane
	float r, r_, f;
	float ndotv;				// N*V (normal dot viewing direction) dotproduct
	float sx, sy, tx, ty;		// derivatives of the screen to ST plane mapping
	float m11, m12, m22;
	float a, b, b_2, c;			// the EWA ellipse coefficients

	int Xmin, Xmax, Ymin, Ymax;	// bounding box of the ellipse to be rasterized
	float lx, ly;
	int X, Y;
	float x, y;
	float q, dq, ddq;

	float dzc_dxs, dzc_dys;			// dzc/dxs, dzc/dys derivatives
	float z_start, z_cur;			// z values to be rasterized

	int i;
	float r_comp, g_comp, b_comp;

	float threshold_c3;
	float e;
	float w;

	float l_2, _l_2;
	float det_;

	Vector3D normal;
	float _radius;

	//scale z buffer according to surfel radius
	float scale_otoc_radius;

	// init bounding box return value
	bbox[0] = width;
	bbox[1] = height;
	bbox[2] = bbox[3] = 0;

	l_2 = (float)(1 << l);
	_l_2 = 1 / l_2;

	// thresholds for the 'merge/separate' decision
	// note: 
	// - these thresholds are dependent on the level of the warped samples, since the
	// distance on the surface between samples increases with the warped level.
	// - these offsets should be constant in units 'dz per unit in object space'. but since 
	// z coordinates in the z-buffer are in camera coordinates, we have to transform the offsets
	// to camera space by applying the 'scale_otoc' (object to camera space) scaling
	threshold_c3 = zbf_angleThreshold;							// cut off for 'ndotv' used for calculating the derivatives dzc_dxs,dzc_dys

	l_2 *= l_2;

	// step 1: calculate the EWA ellipse coefficients

	// compute normalized viewing vector V
	// restore camera coordinates of projected point (on z=1 plane)
	V_x = -(x0 / vp_sx - vp_tx);
	V_y = -(y0 / vp_sy - vp_ty);
	V_z = -1.f;
	r_ = 1.f / (float)sqrt(V_x * V_x + V_y * V_y + 1.f);
	V_x *= r_; V_y *= r_; V_z *= r_;
	ndotv = n[0] * V_x + n[1] * V_y + n[2] * V_z;

	if (ndotv < 0) {
		n[0] = -n[0];
		n[1] = -n[1];
		n[2] = -n[2];
		ndotv = -ndotv;
	}

	// project screen x,y unit vectors along V onto ST plane
	// determine direction Ix,Iy of projection of x,y
	Ix_x = -V_z * n[2] - V_y * n[1];		// (X x V) x N
	Ix_y = V_y * n[0];
	Ix_z = V_z * n[0];
	Iy_x = V_x * n[1];				// (Y x V) x N
	Iy_y = -V_x * n[0] - V_z * n[2];
	Iy_z = V_z * n[1];

	// f given by the vector equation Y+g*V=f*Iy resp X+g*V=f*Ix
	// Iy*f resp. Ix*f is the intersection point of Iy,Ix with Y+g*V resp. X+g*V
	// (see SurfaceSplatting.mws)
	f = -1.f / ndotv;

	// dzdx, dzdy derivatives for rasterizing z values on the ellipse
	if (f < -threshold_c3) e = -threshold_c3; else e = f;
	// note: since z coordinates in the z buffer are camera coordinate z-values, we have
	// to transform from screen to camera coordinates (by concatenating the screen-to-object
	// and object-to-camera scalings).
	dzc_dxs = Ix_z * e * scale_stoo * scale_otoc * z;
	dzc_dys = Iy_z * e * scale_stoo * scale_otoc * z;
	e = -e;

	// normalize Ix
	r = (float)sqrt(Ix_x * Ix_x + Ix_y * Ix_y + Ix_z * Ix_z);
	r_ = 1 / r;
	S_x = Ix_x * r_;
	S_y = Ix_y * r_;
	S_z = Ix_z * r_;

	// determine T parameter direction on ST plane
	// note: S parameter direction is chosen as Ix
	// T is automatically normalized,  since n and S have unit length
	// and are orthogonal by construction of S
	T_x = n[1] * S_z - n[2] * S_y;		// N x S
	T_y = n[2] * S_x - n[0] * S_z;
	T_z = n[0] * S_y - n[1] * S_x;

	// compute sx, sy, tx, ty derivatives
	// these derivatives build the inverse jacobian inv(J)=[sx,sy | tx,ty]
	// of the mapping J from object surface to screen

	_radius = surfel->radius;
	scale_otoc_radius = scale_otoc * _radius;
	_radius = 1.0f / _radius;

	Iy_x *= f; Iy_y *= f; Iy_z *= f;		// f*Iy is the intersection point with Y+g*V
	sx = r * f * scale_stoo * z * _radius * _l_2;		// note f given by the vector equation X+g*V=f*Ix takes the same value as above
	tx = 0;
	sy = (Iy_x * S_x + Iy_y * S_y + Iy_z * S_z) * scale_stoo * z * _radius * _l_2;		// Iy*S projects Iy onto S
	ty = (Iy_x * T_x + Iy_y * T_y + Iy_z * T_z) * scale_stoo * z * _radius * _l_2;		// Iy*T projects Iy onto T

	// compute z-range of the reconstruction kernel
	// see ellipseboundingbox_general.mws, idea: compute the point on the ellipse, where the ellipse tangent
	// is perpendicular to the depth gradient, i.e. [dzc_dxs, dzc_dys]. this is the point with maximum depth
	// on the ellipse
	// NOTE: to avoid overshoots in z, we use the z-range of the reconstruction kernel for blending!
	// NOTE: the variable "d" in the maple sheet corresponds to "-zbf_cutoffRadius_2"!
	// first, compute the conic matrix of the reconstruction kernel, which is [a b/2 | b/2 c] = inv(J)^T*inv(J)
	a = sx * sx;
	b_2 = sx * sy;
	b = 2.f * b_2;
	c = sy * sy + ty * ty;
	float discr;
	discr = -4 * a * dzc_dxs * b * dzc_dys * c - a * dzc_dys * dzc_dys * b * b + 4 * c * dzc_dys * dzc_dys * a * a +
		4 * a * dzc_dxs * dzc_dxs * c * c + b * b * b * dzc_dxs * dzc_dys - b * b * dzc_dxs * dzc_dxs * c;

	float zExtremum_x, zExtremum_y, tmp;
	tmp = (float)sqrt(discr * zbf_cutoffRadius_2);
	zExtremum_x = tmp * (-dzc_dys * b + 2 * dzc_dxs * c) / discr;
	zExtremum_y = tmp * (-2 * dzc_dys * a + dzc_dxs * b) / discr;

	float zRange_x, zRange_y;
	tmp = zExtremum_x * dzc_dxs;
	zRange_x = (tmp < 0) ? -tmp : tmp;
	tmp = zExtremum_y * dzc_dys;
	zRange_y = (tmp < 0) ? -tmp : tmp;

	float zMax, zMin;
	zMax = z + zRange_x + zRange_y;
	zMin = z - zRange_x - zRange_y;

	// guarantee a certain minimum z-range, otherwise blending fails for splats exactly parallel to the
	// image plane (the minimum z-range is 1 in object space, and then scaled to camera space).
	if (zMax - zMin < l_2 * scale_otoc_radius) {
		zMax += 0.5f * l_2 * scale_otoc_radius;
		zMin -= 0.5f * l_2 * scale_otoc_radius;
	}

	// calculate the matrix e[]=inv(J*transpose(J)+I), which describes the
	// EWA ellipse on the screen
	// note: column vectors are used, therefore the calculations are slightly
	// different to Heckbert's notation (cf. Maple file SurfaceSplatting2.mws)

	// the filtered variance matrix m[]
	// m[] = J*transpose(J)+I (details see SurfaceSplatting2.mws)
	r_ = 1.f / (sx * ty);
	r_ *= r_;							// r_ = 1/(sx*ty)^2, note that always tx=0, therefore some terms are missing
	m11 = (ty * ty + sy * sy) * r_ + 1.0f;	// in these formulas compared to the maple sheet
	m12 = -sy * sx * r_;
	m22 = sx * sx * r_ + 1.0f;

	// the filtered conic matric m[]^(-1)
	// matrix(A,B/2,B/2,C) = inv(m[])
	r_ = 1.f / (m11 * m22 - m12 * m12);
	a = m22 * r_;
	b_2 = -m12 * r_;
	b = b_2 * 2.f;
	c = m11 * r_;

	// calculate the normalization factor for the gaussian filter, which corresponds to the 
	// area of the reconstruction filter in source space, thus it is 
	// 1 / sqrt( det(inv(J)*transpose(inv(J))+I) )
	// note: the factor l_2 compensate for the level of detail of the LDC tree
	det_ = l_2 / (float)sqrt(sx * sx * ty * ty + sx * sx + sy * sy + ty * ty + 1.f);

	// bounding box of the ellipse
	// see ellipseboundingbox.mws, an exact axis aligned bounding box is computed by finding the points on
	// the ellipse where the tangent of the ellipse is parallel to x- and y-axis respectively.
	// NOTE: the variable "d" in the maple sheet corresponds to "-zbf_cutoffRadius_2"!
	discr = (float)sqrt((-b * b + 4 * c * a) * zbf_cutoffRadius_2 * a);
	ly = 2.f / (-b * b + 4 * c * a) * discr;

	discr = (float)sqrt(c * (-b * b + 4 * c * a) * zbf_cutoffRadius_2);
	lx = 2.f / (-b * b + 4 * c * a) * discr;

	lx = (lx < 0) ? -lx : lx;
	ly = (ly < 0) ? -ly : ly;
	Xmax = (int)(x0 + lx) + 1;
	Xmin = (int)(x0 - lx);
	Ymax = (int)(y0 + ly) + 1;
	Ymin = (int)(y0 - ly);

	// get surfel color
	r_comp = surfel->r;
	g_comp = surfel->g;
	b_comp = surfel->b;

	// step 2: rasterize the EWA ellipse

	// padding
	if (Xmin < 0) {
		Xmin = 0;
		if (Xmax < 0)
			return;
	}
	if (Xmax >= width) {
		Xmax = width - 1;
		if (Xmin >= width)
			return;
	}
	if (Ymin < 0) {
		Ymin = 0;
		if (Ymax < 0)
			return;
	}
	if (Ymax >= height) {
		Ymax = height - 1;
		if (Ymin >= height)
			return;
	}

	// set bounding box (the bounding box is a return value)
	bbox[0] = Xmin;
	bbox[1] = Ymin;
	bbox[2] = Xmax;
	bbox[3] = Ymax;

	// z value in the lower left corner of the rasterized area
	z_start = z - dzc_dxs * (x0 - ((float)Xmin + 0.5f)) - dzc_dys * (y0 - ((float)Ymin + 0.5f));

	x = ((float)Xmin + 0.5f) - x0;
	ddq = 2 * a;

	// *********************
	// ellipse rasterization
	// *********************
	for (Y = Ymin; Y <= Ymax; Y++)
	{
		// finite differences for ellipse rasterization
		y = ((float)Y + 0.5f) - y0;
		dq = a * (2 * x + 1) + b * y;
		q = (c * y + b * x) * y + a * x * x;

		// init z value
		z_cur = z_start;

		for (X = Xmin; X <= Xmax; X++)
		{
			i = X + width * Y;

			if (q < zbf_cutoffRadius_2) {
				if (zMin <= zBuffer[i].zMax)
				{
					// merge contributions
					w = filterLUT[(int)(q * _zbf_cutoffRadius_2 * zbf_LUTsize)] * det_;

					atomicAdd(&zBuffer[i].w, w);

					// add color contribution
					atomicAdd(&zBuffer[i].c[0], r_comp * w);
					atomicAdd(&zBuffer[i].c[1], g_comp * w);
					atomicAdd(&zBuffer[i].c[2], b_comp * w);

					// normals
					atomicAdd(&zBuffer[i].n[0], n[0] * w);
					atomicAdd(&zBuffer[i].n[1], n[1] * w);
					atomicAdd(&zBuffer[i].n[2], n[2] * w);
				}
			}
			q += dq;
			dq += ddq;

			z_cur += dzc_dxs;
		}
		z_start += dzc_dys;
	}
}

__device__ void projectSampleStep1GPU(int width, int height, Warper* warper, ZBufferProperty* zBufferProperty, ZBufferItem* zBuffer, float* filterLUT, Surfel* surfel, int* bbox)
{
	int i, j;

	float wrp_frustum_nearplane;
	float wrp_frustum_farplane;
	float vp_sx, vp_sy;				// scaling for viewport mapping
	float vp_tx[3];							// translation for viewport mapping, x direction
	float vp_ty[3];							// translation for viewport mapping, y direction
	float A[3][9];					// the transformation matrix (in 3 variants, one for each base plane)
	float v[3];						// the translation vector
	float normalsA[9];				// the transposed inverse of A for transforming normals from camera to object space
	float stoo_scaling;				// screen to camera scaling due to viewport mapping and uniform scaling in the modelview transform
	float otoc_scaling;				// object to camera space scaling, due to scaling in transformation matrix

	// static variables used for warping, which are independent of current block
	wrp_frustum_nearplane = warper->frustum.nearplane;
	wrp_frustum_farplane = warper->frustum.farplane;
	stoo_scaling = warper->frustum.xP * 2 / (width * warper->transformation.scaling);
	otoc_scaling = warper->transformation.scaling;

	// set transformation variables
	for (i = 0; i < 9; i++)
		A[0][i] = normalsA[i] = warper->transformation.rotation[i];
	for (i = 0; i < 3; i++) v[i] = warper->transformation.translation[i];

	// rotation matrices for yz- and zx-baseplanes
	for (i = 0; i < 3; i++)
	{
		for (j = 0; j < 3; j++)
		{
			A[1][i * 3 + j] = A[0][i * 3 + (j + 1) % 3];
			A[2][i * 3 + j] = A[0][i * 3 + (j + 2) % 3];
		}
	}

	// set viewport mapping variables
	vp_sx = width / (2 * warper->frustum.xP);
	vp_sy = height / (2 * warper->frustum.yP);

	// set variables for warping from all 3 baseplanes
	for (i = 0; i < 3; i++)
	{
		vp_tx[i] = warper->frustum.xP - warper->frustum.xC;
		vp_ty[i] = warper->frustum.yP - warper->frustum.yC;
	}

	float x_c, y_c;             // camera-space x and y values
	float z_c, r_z_c;			// camera-space z-value (and its reciprocal) of sample being warped

	float xImg, yImg;			// x- and y-screen-coordinates of warped sample
	float xPad, yPad;			// pads in x and y direction for correct clipping

	Vector3D pos;				// object space sample position
	Vector3D nrm;				// object space normal
	Vector3D tan1,              // object space tangent axes
		tan2;
	float r;					// surfel radius
	float n[3];					// camera space normal
	int splatBBox[4];			// bounding box of rasterized splat

	// get sample position
	pos = surfel->position;

	// apply transformation matrix
	z_c = A[0][6] * pos[0] + A[0][7] * pos[1] + A[0][8] * pos[2] + v[2];
	// apply near and far clipping planes
	if (z_c > wrp_frustum_nearplane && z_c < wrp_frustum_farplane) {

		x_c = A[0][0] * pos[0] + A[0][1] * pos[1] + A[0][2] * pos[2] + v[0];
		y_c = A[0][3] * pos[0] + A[0][4] * pos[1] + A[0][5] * pos[2] + v[1];

		// perspective divide and viewport transformation
		r_z_c = 1 / z_c;
		xImg = (x_c * r_z_c + vp_tx[0]) * vp_sx;
		yImg = (y_c * r_z_c + vp_ty[0]) * vp_sy;

		// for correct clipping: project surfel radius to screen space
		r = surfel->radius;
		r *= 1 / z_c;
		xPad = r * vp_sx;
		yPad = r * vp_sy;

		// put it into the z-buffer
		if ((xImg >= -xPad) && (xImg < width + xPad) &&
			(yImg >= -yPad) && (yImg < height + yPad))
		{
			// transform normal to camera coordinates
			nrm = surfel->normal;

			n[0] = normalsA[0] * nrm[0] + normalsA[1] * nrm[1] + normalsA[2] * nrm[2];
			n[1] = normalsA[3] * nrm[0] + normalsA[4] * nrm[1] + normalsA[5] * nrm[2];
			n[2] = normalsA[6] * nrm[0] + normalsA[7] * nrm[1] + normalsA[8] * nrm[2];

			// caution: this function (or macro) relies on global variables!
			// note: 'warped level' is set to 0
			surfaceSplatStep1GPU(width, height, zBufferProperty, zBuffer, filterLUT, xImg, yImg, z_c, n, surfel, 0, stoo_scaling, otoc_scaling, vp_sx, vp_sy, vp_tx[0], vp_ty[0], splatBBox);

			// enlarge bounding box (containing the modified frame buffer area) if necessary
			atomicMin(&bbox[0], splatBBox[0]);
			atomicMin(&bbox[1], splatBBox[1]);
			atomicMax(&bbox[2], splatBBox[2]);
			atomicMax(&bbox[3], splatBBox[3]);
		}
	}
}

__device__ void projectSampleStep2GPU(int width, int height, Warper* warper, ZBufferProperty* zBufferProperty, ZBufferItem* zBuffer, float* filterLUT, Surfel* surfel, int* bbox)
{
	int i, j;

	float wrp_frustum_nearplane;
	float wrp_frustum_farplane;
	float vp_sx, vp_sy;				// scaling for viewport mapping
	float vp_tx[3];							// translation for viewport mapping, x direction
	float vp_ty[3];							// translation for viewport mapping, y direction
	float A[3][9];					// the transformation matrix (in 3 variants, one for each base plane)
	float v[3];						// the translation vector
	float normalsA[9];				// the transposed inverse of A for transforming normals from camera to object space
	float stoo_scaling;				// screen to camera scaling due to viewport mapping and uniform scaling in the modelview transform
	float otoc_scaling;				// object to camera space scaling, due to scaling in transformation matrix

	// static variables used for warping, which are independent of current block
	wrp_frustum_nearplane = warper->frustum.nearplane;
	wrp_frustum_farplane = warper->frustum.farplane;
	stoo_scaling = warper->frustum.xP * 2 / (width * warper->transformation.scaling);
	otoc_scaling = warper->transformation.scaling;

	// set transformation variables
	for (i = 0; i < 9; i++)
		A[0][i] = normalsA[i] = warper->transformation.rotation[i];
	for (i = 0; i < 3; i++) v[i] = warper->transformation.translation[i];

	// rotation matrices for yz- and zx-baseplanes
	for (i = 0; i < 3; i++)
	{
		for (j = 0; j < 3; j++)
		{
			A[1][i * 3 + j] = A[0][i * 3 + (j + 1) % 3];
			A[2][i * 3 + j] = A[0][i * 3 + (j + 2) % 3];
		}
	}

	// set viewport mapping variables
	vp_sx = width / (2 * warper->frustum.xP);
	vp_sy = height / (2 * warper->frustum.yP);

	// set variables for warping from all 3 baseplanes
	for (i = 0; i < 3; i++)
	{
		vp_tx[i] = warper->frustum.xP - warper->frustum.xC;
		vp_ty[i] = warper->frustum.yP - warper->frustum.yC;
	}

	float x_c, y_c;             // camera-space x and y values
	float z_c, r_z_c;			// camera-space z-value (and its reciprocal) of sample being warped

	float xImg, yImg;			// x- and y-screen-coordinates of warped sample
	float xPad, yPad;			// pads in x and y direction for correct clipping

	Vector3D pos;				// object space sample position
	Vector3D nrm;				// object space normal
	Vector3D tan1,              // object space tangent axes
		tan2;
	float r;					// surfel radius
	float n[3];					// camera space normal
	int splatBBox[4];			// bounding box of rasterized splat

	// get sample position
	pos = surfel->position;

	// apply transformation matrix
	z_c = A[0][6] * pos[0] + A[0][7] * pos[1] + A[0][8] * pos[2] + v[2];
	// apply near and far clipping planes
	if (z_c > wrp_frustum_nearplane && z_c < wrp_frustum_farplane) {

		x_c = A[0][0] * pos[0] + A[0][1] * pos[1] + A[0][2] * pos[2] + v[0];
		y_c = A[0][3] * pos[0] + A[0][4] * pos[1] + A[0][5] * pos[2] + v[1];

		// perspective divide and viewport transformation
		r_z_c = 1 / z_c;
		xImg = (x_c * r_z_c + vp_tx[0]) * vp_sx;
		yImg = (y_c * r_z_c + vp_ty[0]) * vp_sy;

		// for correct clipping: project surfel radius to screen space
		r = surfel->radius;
		r *= 1 / z_c;
		xPad = r * vp_sx;
		yPad = r * vp_sy;

		// put it into the z-buffer
		if ((xImg >= -xPad) && (xImg < width + xPad) &&
			(yImg >= -yPad) && (yImg < height + yPad))
		{
			// transform normal to camera coordinates
			nrm = surfel->normal;

			n[0] = normalsA[0] * nrm[0] + normalsA[1] * nrm[1] + normalsA[2] * nrm[2];
			n[1] = normalsA[3] * nrm[0] + normalsA[4] * nrm[1] + normalsA[5] * nrm[2];
			n[2] = normalsA[6] * nrm[0] + normalsA[7] * nrm[1] + normalsA[8] * nrm[2];

			// caution: this function (or macro) relies on global variables!
			// note: 'warped level' is set to 0
			surfaceSplatStep2GPU(width, height, zBufferProperty, zBuffer, filterLUT, xImg, yImg, z_c, n, surfel, 0, stoo_scaling, otoc_scaling, vp_sx, vp_sy, vp_tx[0], vp_ty[0], splatBBox);

			// enlarge bounding box (containing the modified frame buffer area) if necessary
			atomicMin(&bbox[0], splatBBox[0]);
			atomicMin(&bbox[1], splatBBox[1]);
			atomicMax(&bbox[2], splatBBox[2]);
			atomicMax(&bbox[3], splatBBox[3]);
		}
	}
}

__device__ void projectSampleStep3GPU(int width, int height, Warper* warper, ZBufferProperty* zBufferProperty, ZBufferItem* zBuffer, float* filterLUT, Surfel* surfel, int* bbox)
{
	int i, j;

	float wrp_frustum_nearplane;
	float wrp_frustum_farplane;
	float vp_sx, vp_sy;				// scaling for viewport mapping
	float vp_tx[3];							// translation for viewport mapping, x direction
	float vp_ty[3];							// translation for viewport mapping, y direction
	float A[3][9];					// the transformation matrix (in 3 variants, one for each base plane)
	float v[3];						// the translation vector
	float normalsA[9];				// the transposed inverse of A for transforming normals from camera to object space
	float stoo_scaling;				// screen to camera scaling due to viewport mapping and uniform scaling in the modelview transform
	float otoc_scaling;				// object to camera space scaling, due to scaling in transformation matrix

	// static variables used for warping, which are independent of current block
	wrp_frustum_nearplane = warper->frustum.nearplane;
	wrp_frustum_farplane = warper->frustum.farplane;
	stoo_scaling = warper->frustum.xP * 2 / (width * warper->transformation.scaling);
	otoc_scaling = warper->transformation.scaling;

	// set transformation variables
	for (i = 0; i < 9; i++)
		A[0][i] = normalsA[i] = warper->transformation.rotation[i];
	for (i = 0; i < 3; i++) v[i] = warper->transformation.translation[i];

	// rotation matrices for yz- and zx-baseplanes
	for (i = 0; i < 3; i++)
	{
		for (j = 0; j < 3; j++)
		{
			A[1][i * 3 + j] = A[0][i * 3 + (j + 1) % 3];
			A[2][i * 3 + j] = A[0][i * 3 + (j + 2) % 3];
		}
	}

	// set viewport mapping variables
	vp_sx = width / (2 * warper->frustum.xP);
	vp_sy = height / (2 * warper->frustum.yP);

	// set variables for warping from all 3 baseplanes
	for (i = 0; i < 3; i++)
	{
		vp_tx[i] = warper->frustum.xP - warper->frustum.xC;
		vp_ty[i] = warper->frustum.yP - warper->frustum.yC;
	}

	float x_c, y_c;             // camera-space x and y values
	float z_c, r_z_c;			// camera-space z-value (and its reciprocal) of sample being warped

	float xImg, yImg;			// x- and y-screen-coordinates of warped sample
	float xPad, yPad;			// pads in x and y direction for correct clipping

	Vector3D pos;				// object space sample position
	Vector3D nrm;				// object space normal
	Vector3D tan1,              // object space tangent axes
		tan2;
	float r;					// surfel radius
	float n[3];					// camera space normal
	int splatBBox[4];			// bounding box of rasterized splat

	// get sample position
	pos = surfel->position;

	// apply transformation matrix
	z_c = A[0][6] * pos[0] + A[0][7] * pos[1] + A[0][8] * pos[2] + v[2];
	// apply near and far clipping planes
	if (z_c > wrp_frustum_nearplane && z_c < wrp_frustum_farplane) {

		x_c = A[0][0] * pos[0] + A[0][1] * pos[1] + A[0][2] * pos[2] + v[0];
		y_c = A[0][3] * pos[0] + A[0][4] * pos[1] + A[0][5] * pos[2] + v[1];

		// perspective divide and viewport transformation
		r_z_c = 1 / z_c;
		xImg = (x_c * r_z_c + vp_tx[0]) * vp_sx;
		yImg = (y_c * r_z_c + vp_ty[0]) * vp_sy;

		// for correct clipping: project surfel radius to screen space
		r = surfel->radius;
		r *= 1 / z_c;
		xPad = r * vp_sx;
		yPad = r * vp_sy;

		// put it into the z-buffer
		if ((xImg >= -xPad) && (xImg < width + xPad) &&
			(yImg >= -yPad) && (yImg < height + yPad))
		{
			// transform normal to camera coordinates
			nrm = surfel->normal;

			n[0] = normalsA[0] * nrm[0] + normalsA[1] * nrm[1] + normalsA[2] * nrm[2];
			n[1] = normalsA[3] * nrm[0] + normalsA[4] * nrm[1] + normalsA[5] * nrm[2];
			n[2] = normalsA[6] * nrm[0] + normalsA[7] * nrm[1] + normalsA[8] * nrm[2];

			// caution: this function (or macro) relies on global variables!
			// note: 'warped level' is set to 0
			surfaceSplatStep3GPU(width, height, zBufferProperty, zBuffer, filterLUT, xImg, yImg, z_c, n, surfel, 0, stoo_scaling, otoc_scaling, vp_sx, vp_sy, vp_tx[0], vp_ty[0], splatBBox);

			// enlarge bounding box (containing the modified frame buffer area) if necessary
			atomicMin(&bbox[0], splatBBox[0]);
			atomicMin(&bbox[1], splatBBox[1]);
			atomicMax(&bbox[2], splatBBox[2]);
			atomicMax(&bbox[3], splatBBox[3]);
		}
	}
}

__global__ void kernelStep1(int width, int height, Warper* warper, ZBufferProperty* zBufferProperty, ZBufferItem* zBuffer, float* filterLUT, int numSurfels, Surfel* surfels, int* bbox) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	while (i < numSurfels) {
		projectSampleStep1GPU(width, height, warper, zBufferProperty, zBuffer, filterLUT, &surfels[i], bbox);
		i += gridDim.x * blockDim.x;
	}
}

__global__ void kernelStep2(int width, int height, Warper* warper, ZBufferProperty* zBufferProperty, ZBufferItem* zBuffer, float* filterLUT, int numSurfels, Surfel* surfels, int* bbox) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	while (i < numSurfels) {
		projectSampleStep2GPU(width, height, warper, zBufferProperty, zBuffer, filterLUT, &surfels[i], bbox);
		i += gridDim.x * blockDim.x;
	}
}

__global__ void kernelStep3(int width, int height, Warper* warper, ZBufferProperty* zBufferProperty, ZBufferItem* zBuffer, float* filterLUT, int numSurfels, Surfel* surfels, int* bbox) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	while (i < numSurfels) {
		projectSampleStep3GPU(width, height, warper, zBufferProperty, zBuffer, filterLUT, &surfels[i], bbox);
		i += gridDim.x * blockDim.x;
	}
}

void projectGPU(int width, int height, Warper* warper, ZBufferProperty* zBufferProperty, ZBufferItem* zBuffer, float* filterLUT, int numSurfels, Surfel* surfels, int bbox[4]) {
	double t0 = glfwGetTime();

	Warper* warperGpu;
	hipMalloc(&warperGpu, sizeof(Warper));
	hipMemcpy(warperGpu, warper, sizeof(Warper), hipMemcpyHostToDevice);

	int* bboxGpu;
	hipMalloc(&bboxGpu, sizeof(int) * 4);
	hipMemcpy(bboxGpu, bbox, sizeof(int) * 4, hipMemcpyHostToDevice);

	double t1 = glfwGetTime();

	kernelStep1<<<512, 512>>>(width, height, warperGpu, zBufferProperty, zBuffer, filterLUT, numSurfels, surfels, bboxGpu);
	kernelStep2<<<512, 512>>>(width, height, warperGpu, zBufferProperty, zBuffer, filterLUT, numSurfels, surfels, bboxGpu);
	kernelStep3<<<512, 512>>>(width, height, warperGpu, zBufferProperty, zBuffer, filterLUT, numSurfels, surfels, bboxGpu);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));

	double t2 = glfwGetTime();

	//hipMemcpy(zBuffer, zBufferGpu, sizeof(ZBufferItem) * zBufferProperty->bufsize, hipMemcpyDeviceToHost);
	hipMemcpy(bbox, bboxGpu, sizeof(int) * 4, hipMemcpyDeviceToHost);

	double t3 = glfwGetTime();
	//std::cout << t1 - t0 << ' ' << t2 - t1 << ' ' << t3 - t2 << std::endl;

	hipFree(warperGpu);
	hipFree(bboxGpu);
}